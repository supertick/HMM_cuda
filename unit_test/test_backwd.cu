#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>		// check errors
// #include <hipblas.h>		      	// cublas
#include <hipblas.h>	      	// new cublas api: pointer mode

#include "kernel_forward.h"
#include "kernel_backward.h"
#include "kernel_bw.h"

/// cpu parameters
int N = 64, T = 64;
float alp = 1.f;                                                                
float bet = 0.f;
size_t bytes_nt  = sizeof(float) * N * T;                                          
size_t bytes_n   = sizeof(float) * N;
size_t bytes_ll  = sizeof(float) * (T + 1);
size_t bytes_nn  = sizeof(float) * N * N;

float *b;    
float *pi;   
float *alpha;
float *ones;
float *beta;
float *ll;
float *a;
float *betaB;

/// gpu memory
float *b_d; 
float *pi_d; 
float *alpha_d; 
float *ones_d; 
float *beta_d;
float *ll_d;
float *a_d;
float *betaB_d;

/// cublas
hipblasStatus_t ret;                                                               
hipblasHandle_t handle, handle1;

/// allocate resource
void allocate_data()
{
	/// cpu
	b      = (float *) malloc (bytes_nt);
	pi     = (float *) malloc (bytes_n);
	alpha  = (float *) malloc (bytes_nt);
	ones   = (float *) malloc (bytes_n);
	beta   = (float *) malloc (bytes_nt);
	ll     = (float *) malloc (bytes_ll);
	a      = (float *) malloc (bytes_nn);
	betaB  = (float *) malloc (bytes_n);

	/// gpu
	checkCudaErrors(hipMalloc((void**)&b_d,     bytes_nt));                        
	checkCudaErrors(hipMalloc((void**)&pi_d,    bytes_n));     
	checkCudaErrors(hipMalloc((void**)&alpha_d, bytes_nt));
	checkCudaErrors(hipMalloc((void**)&ones_d,  bytes_n));     
	checkCudaErrors(hipMalloc((void**)&beta_d,  bytes_nt));
	checkCudaErrors(hipMalloc((void**)&ll_d,    bytes_ll));
	checkCudaErrors(hipMalloc((void**)&a_d,     bytes_nn));
	checkCudaErrors(hipMalloc((void**)&betaB_d, bytes_n));  

	/// cublas
	// ret = cublasInit();                                                         
	// if (ret != HIPBLAS_STATUS_SUCCESS) {
	// 	fprintf (stderr, "ERROR: CUBLAS Initialization failure\n");             
	// 	exit(EXIT_FAILURE);                                                     
	// }                                                                           
                                                                                
    ret  = hipblasCreate(&handle);
    ret  = hipblasCreate(&handle1);

	hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
}

/// release
void release_data()
{
	/// cpu
	free(b);
	free(pi);
	free(alpha);
	free(ones);
	free(beta);
	free(ll);
	free(a);
	free(betaB);


	/// gpu
	checkCudaErrors(hipFree(b_d));
	checkCudaErrors(hipFree(pi_d));
	checkCudaErrors(hipFree(alpha_d));
	checkCudaErrors(hipFree(ones_d));
	checkCudaErrors(hipFree(beta_d));
	checkCudaErrors(hipFree(ll_d));
	checkCudaErrors(hipFree(a_d));
	checkCudaErrors(hipFree(betaB_d));
}



TEST(HMM_Backward, update_beta) 
{
	/// allocate
	allocate_data();

	/// configure cpu data
	for (int i = 0; i < (N * T); i++) {                                             
		b[i]    = 0.15f;                                                   
		beta[i] = 0.2f;                                                   
	}

	/// host to device
    checkCudaErrors(hipMemcpy(b_d,    b,    bytes_nt, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(beta_d, beta, bytes_nt, hipMemcpyHostToDevice));

	// kernel
	bk_update_beta (&beta_d[0], &b_d[0], betaB_d, N);

	/// device to host
    checkCudaErrors(hipMemcpy(betaB, betaB_d, bytes_n, hipMemcpyDeviceToHost));

	/// check results
	int expect = 0;
	for (int i = 0; i < N; i++) {                                             
		if (abs(betaB[i] - 0.03f) >= 1e-5) {
			printf("betaB[%d] = %f\n", i, betaB[i]);
			expect |= 1;		
			break;
		}
	}

	EXPECT_EQ(0, expect);

	/// release
	release_data();
}



TEST(HMM_Backward, scale_beta) 
{
	/// allocate
	allocate_data();

	/// configure cpu data
	for (int i = 0; i < (N * T); i++) {                                             
		beta[i] = 2.f;                                                   
	}

	ll[0] = 4.f; 

	/// host to device
    checkCudaErrors(hipMemcpy(ll_d,   ll,   bytes_ll, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(beta_d, beta, bytes_nt, hipMemcpyHostToDevice));

	// kernel
	bk_scaling (N, &beta_d[0], ll_d); 

	/// device to host
    checkCudaErrors(hipMemcpy(beta, beta_d, bytes_nt, hipMemcpyDeviceToHost));

	/// check results
	int expect = 0;
	for (int i = 0; i < N; i++) {                                             
		if (abs(beta[i] - 0.5f) >= 1e-5) {
			printf("beta[%d] = %f\n", i, beta[i]);
			expect |= 1;		
			break;
		}
	}

	EXPECT_EQ(0, expect);

	/// release
	release_data();
}
