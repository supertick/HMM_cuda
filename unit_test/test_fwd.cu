#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>		// check errors
// #include <hipblas.h>		      	// cublas
#include <hipblas.h>	      	// new cublas api: pointer mode

#include "kernel_forward.h"
#include "kernel_backward.h"
#include "kernel_bw.h"

/// cpu parameters
int N = 64, T = 64;
float alp = 1.f;                                                                
float bet = 0.f;
size_t bytes_nt  = sizeof(float) * N * T;                                          
size_t bytes_n   = sizeof(float) * N;
size_t bytes_ll  = sizeof(float) * (T + 1);
size_t bytes_nn  = sizeof(float) * N * N;

float *b;    
float *pi;   
float *alpha;
float *ones;
float *beta;
float *ll;
float *a;

/// gpu memory
float *b_d; 
float *pi_d; 
float *alpha_d; 
float *ones_d; 
float *beta_d;
float *ll_d;
float *a_d;

/// cublas
hipblasStatus_t ret;                                                               
hipblasHandle_t handle, handle1;

/// allocate resource
void allocate_data()
{
	/// cpu
	b      = (float *) malloc (bytes_nt);
	pi     = (float *) malloc (bytes_n);
	alpha  = (float *) malloc (bytes_nt);
	ones   = (float *) malloc (bytes_n);
	beta   = (float *) malloc (bytes_nt);
	ll     = (float *) malloc (bytes_ll);
	a      = (float *) malloc (bytes_nn);

	/// gpu
	checkCudaErrors(hipMalloc((void**)&b_d,     bytes_nt));                        
	checkCudaErrors(hipMalloc((void**)&pi_d,    bytes_n));     
	checkCudaErrors(hipMalloc((void**)&alpha_d, bytes_nt));
	checkCudaErrors(hipMalloc((void**)&ones_d,  bytes_n));     
	checkCudaErrors(hipMalloc((void**)&beta_d,  bytes_nt));
	checkCudaErrors(hipMalloc((void**)&ll_d,    bytes_ll));
	checkCudaErrors(hipMalloc((void**)&a_d,     bytes_nn));

	/// cublas
	// ret = cublasInit();                                                         
	// if (ret != HIPBLAS_STATUS_SUCCESS) {
	// 	fprintf (stderr, "ERROR: CUBLAS Initialization failure\n");             
	// 	exit(EXIT_FAILURE);                                                     
	// }                                                                           
                                                                                
    ret  = hipblasCreate(&handle);
    ret  = hipblasCreate(&handle1);

	hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
}

/// release
void release_data()
{
	/// cpu
	free(b);
	free(pi);
	free(alpha);
	free(ones);
	free(beta);
	free(ll);
	free(a);


	/// gpu
	checkCudaErrors(hipFree(b_d));
	checkCudaErrors(hipFree(pi_d));
	checkCudaErrors(hipFree(alpha_d));
	checkCudaErrors(hipFree(ones_d));
	checkCudaErrors(hipFree(beta_d));
	checkCudaErrors(hipFree(ll_d));
	checkCudaErrors(hipFree(a_d));
}



/// forward algo: initialize step
TEST(HMM_Forward, intialization) 
{
	/// allocate
	allocate_data();

	/// configure cpu data
	for (int i = 0; i < (N * T); i++) {                                             
		b[i] = 0.3f;                                                   
	}

	for (int i = 0; i < N; i++) {                                                   
		pi[i] = 0.3f;                                                  
	}     

	/// host to device
    checkCudaErrors(hipMemcpy(b_d,  b,  bytes_nt, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(pi_d, pi, bytes_n,  hipMemcpyHostToDevice));


	/// alpha = b * pi
	/// initialize ones_d for cublas
	/// initialize beta_d
	fwd_init_alpha (b_d,                                                        
			pi_d,                                                       
			N,                                                          
			&alpha_d[0],		// the first N samples
			ones_d,                                                     
			&beta_d[(T-1)*N]);  

	/// device to host
    checkCudaErrors(hipMemcpy(alpha, alpha_d, bytes_nt, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(ones, ones_d, bytes_n, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(beta, beta_d, bytes_nt, hipMemcpyDeviceToHost));

	/// check results
	int expect = 0;

	/// check the first N samples of alpha_d
	for (int i = 0; i < N; i++) {                                             
		/// if the difference is larger than 0.00001, trigger the alarm 
		if (abs(alpha[i] - 0.09f) >= 1e-5) {
			expect |= 1;		
			break;
		}
	}

	/// check ones_d
	for (int i = 0; i < N; i++) {                                             
		/// if the difference is larger than 0.00001, trigger the alarm 
		if (abs(ones[i] - 1.f) >= 1e-5) {
			expect |= 1;		
			break;
		}
	}

	/// check beta_d
	for (int i = (T-1) * N; i < T * N; i++) {                                             
		/// if the difference is larger than 0.00001, trigger the alarm 
		if (abs(beta[i] - 1.f) >= 1e-5) {
			expect |= 1;		
			break;
		}
	}


	EXPECT_EQ(0, expect);

	/// release
	release_data();
}



// test cublas_sdot
TEST(HMM_Forward, sum_alpha) 
{
	/// allocate
	allocate_data();

	/// configure cpu data
	for (int i = 0; i < (N * T); i++) {                                             
		alpha[i] = 0.3f;                                                   
	}

	for (int i = 0; i < N; i++) {                                                   
		ones[i] = 1.0f;                                                  
	}     

	/// host to device
    checkCudaErrors(hipMemcpy(alpha_d,  alpha, bytes_nt, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(ones_d,   ones,  bytes_n,  hipMemcpyHostToDevice));

	/// sum(alpha)                                                               
	ret = hipblasSdot(handle, N, &alpha_d[0], 1, ones_d, 1, &ll_d[0]);           
	if (ret != HIPBLAS_STATUS_SUCCESS) {
		fprintf (stderr, "ERROR: Sdot execution error. This is line %d.\n", __LINE__);
		exit(EXIT_FAILURE);                                                     
	}    

	/// device to host
    checkCudaErrors(hipMemcpy(ll, ll_d, bytes_ll, hipMemcpyDeviceToHost));
	
	/// check results
	int expect = 0;

	if (abs(ll[0] - 19.2f) >= 1e-5) {
		expect = 1;		
	}

	// printf("ll[0] = %f\n", ll[0]);

	EXPECT_EQ(0, expect);

	/// release
	release_data();
}


// parallel division 
TEST(HMM_Forward, scaling_alpha) 
{
	/// allocate
	allocate_data();

	/// configure cpu data
	for (int i = 0; i < (N * T); i++) {                                             
		alpha[i] = 0.3f;                                                   
	}

	ll[0] = 2.0f;                                                  

	/// host to device
    checkCudaErrors(hipMemcpy(alpha_d,  alpha, bytes_nt,  hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(ll_d,     ll,    bytes_ll,  hipMemcpyHostToDevice));

	/// kernel
	fwd_scaling (N , &alpha_d[0], ll_d, 0);

	/// device to host
    checkCudaErrors(hipMemcpy(alpha, alpha_d, bytes_nt, hipMemcpyDeviceToHost));
	
	/// check results
	int expect = 0;

	for (int i = 0; i < N; i++) {                                             
		if (abs(alpha[i] - 0.15f) >= 1e-5) {
			printf("alpha[%d] = %f\n", i, alpha[i]);
			expect |= 1;		
			break;
		}
	}
	EXPECT_EQ(0, expect);

	/// release
	release_data();
}


/// test a' * alpha
TEST(HMM_Forward, aT_mul_alpha) 
{
	/// allocate
	allocate_data();

	/// configure cpu data
	for (int i = 0; i < (N * N); i++) {                                             
		a[i] = 0.4f;                                                   
	}

	/// 1st column to 0.2f
	for (int row = 0; row < N ; row++) {                                             
		a[row * N] = 0.2f;                                                   
	}

	for (int i = 0; i < N; i++) {                                                   
		alpha[i] = 1.0f;                                                  
	}     

	/// host to device
    checkCudaErrors(hipMemcpy(a_d,     a,     bytes_nn, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(alpha_d, alpha, bytes_nt,  hipMemcpyHostToDevice));

	// a' * alpha                                                           
	// auto transposed due to the column major thing                        
	ret = hipblasSgemv(handle1, HIPBLAS_OP_N,                                 
			N, N,                                                           
			&alp,                                                           
			a_d, N,                                                         
			&alpha_d[0], 1,                                          
			&bet,                                                           
			&alpha_d[N], 1);                                          

	if (ret != HIPBLAS_STATUS_SUCCESS) {
		fprintf (stderr, "ERROR: Sgemv execution error. This is line %d.\n", __LINE__);
		exit(EXIT_FAILURE);                                                 
	}  

	/// device to host
    checkCudaErrors(hipMemcpy(alpha, alpha_d, bytes_nt, hipMemcpyDeviceToHost));
	
	/// check results
	int expect = 0;

	// alpha[N] = 12.8; the others should be 25.6;
	for (int i = 0; i < N; i++) {                                             
		if(i == 0)
		{
			if (abs(alpha[N + i] - 12.8f) >= 1e-5) {
				printf("alpha[%d] = %f\n", N + i, alpha[N + i]);
				expect |= 1;		
				break;
			}
		}
		else
		{
			if (abs(alpha[N + i] - 25.6f) >= 1e-5) {
				printf("alpha[%d] = %f\n", N + i, alpha[N + i]);
				expect |= 1;		
				break;
			}
		}
	}

	EXPECT_EQ(0, expect);

	/// release
	release_data();
}



/// b * (a' * alpha) 
TEST(HMM_Forward, update_alpha) 
{
	/// allocate
	allocate_data();

	/// configure cpu data
	for (int i = 0; i < (N * T); i++) {                                             
		alpha[i] = 0.3f;                                                   
		b[i]     = 0.2f;                                                   
	}

	/// host to device
    checkCudaErrors(hipMemcpy(alpha_d, alpha, bytes_nt,  hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(b_d,     b,     bytes_nt,  hipMemcpyHostToDevice));

	/// kernel
	/// b * (a' * alpha)                                                     
	fwd_calc_alpha (N, &alpha_d[0], &b_d[0]);   

	/// device to host
    checkCudaErrors(hipMemcpy(alpha, alpha_d, bytes_nt, hipMemcpyDeviceToHost));
	
	/// check results
	int expect = 0;

	for (int i = 0; i < N; i++) {                                             
		if (abs(alpha[i] - 0.06f) >= 1e-5) {
			printf("alpha[%d] = %f\n", i, alpha[i]);
			expect |= 1;		
			break;
		}
	}

	EXPECT_EQ(0, expect);

	/// release
	release_data();
}


/// sum up loglikelihood
TEST(HMM_Forward, sum_log_likelihood) 
{
	/// allocate
	allocate_data();

	/// configure cpu data
	for (int i = 0; i < T; i++) {                                             
		ll[i] = 10.f;                                                   
	}

	/// host to device
    checkCudaErrors(hipMemcpy(ll_d, ll, bytes_ll,  hipMemcpyHostToDevice));

	/// kernel
	fwd_sum_ll (T, ll_d);  

	/// device to host
    checkCudaErrors(hipMemcpy(ll, ll_d, bytes_ll, hipMemcpyDeviceToHost));
	
	/// check results
	int expect = 0;

	if (abs(ll[T] - 64.f) >= 1e-5) {
		printf("log10(ll)= %f\n", ll[T]);
		expect = 1;		
	}

	EXPECT_EQ(0, expect);

	/// release
	release_data();
}


